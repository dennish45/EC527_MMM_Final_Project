// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// from https://github.com/sol-prog/cuda_cublas_curand_thrust
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

#ifndef ARR_SIZE
#define ARR_SIZE 8256
#endif

float getChecksum(float* C, int length) {
	int i, j;
	float sum = 0;	

        for (i = 0; i < length; i++) {
                for (j = 0; j < length; j++) {
                        sum += C[i*length+j];
                }
        }

        return sum;
}

void initializeArrayOrdered2D(float *arr, int len) {
  long int i;


  for (i = 0; i < len*len; i++) {
    arr[i] = (float)i;
  }
}

/*
// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	curandGenerator_t prng;
	curandCreateGenerator(&prng, CURAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	curandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}
*/

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// GPU Timing variables
	hipEvent_t startOuter, stopOuter;
	float elapsed_gpu_outer;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = ARR_SIZE;
	
	float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

        initializeArrayOrdered2D(h_A, nr_rows_A);
        initializeArrayOrdered2D(h_B, nr_rows_B);

	hipEventCreate(&startOuter);
	hipEventCreate(&stopOuter);
	// Record event on the default stream
	hipEventRecord(startOuter, 0);

	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

	// If you already have useful values in A and B you can copy them in GPU:
	hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice);

	// Fill the arrays A and B on GPU with random numbers
	//GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	//GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);


	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
	//std::cout << "A =" << std::endl;
	//print_matrix(h_A, nr_rows_A, nr_cols_A);
	//std::cout << "B =" << std::endl;
	//print_matrix(h_B, nr_rows_B, nr_cols_B);

	// Multiply A and B on GPU
	gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

	// Copy (and print) the result on host memory
	hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
	//std::cout << "C =" << std::endl;
	//print_matrix(h_C, nr_rows_C, nr_cols_C);

	// Stop and destroy the timer
	hipEventRecord(stopOuter,0);
	hipEventSynchronize(stopOuter);
	hipEventElapsedTime(&elapsed_gpu_outer, startOuter, stopOuter);
	printf("%f", elapsed_gpu_outer/1000.0);
	hipEventDestroy(startOuter);
	hipEventDestroy(stopOuter);

	float checksum = getChecksum(h_C, ARR_SIZE);

	fprintf(stderr, "Checksum: %f\n", checksum);


	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
